//genCudaFuncts.cu
//Scott Grauer-Gray
//July 2, 2009
//Defines the general CUDA functions

#include "genCudaFuncts.cuh"

//functions are C, so need extern
extern "C"
{

	//function to initialize CUDA
	void cudaInit(int argc, char **argv)
	{
		CUT_DEVICE_INIT(argc, argv);
	}

	//function to exit CUDA
	void cudaExit(int argc, char **argv)
	{
		CUT_EXIT(argc, argv);
	}


	//function to allocate an array of data on the device
	void allocateArray(void **devPtr, int size)
	{
		hipMalloc(devPtr, size);
	}


	//function to free an array of data on the device
	void freeArray(void *devPtr)
	{
		hipFree(devPtr);
	}

	//function to synchronize the threads
	void threadSync()
	{
		hipDeviceSynchronize();
	}

	//function to copy an array of data from the device to the host
	void copyArrayFromDevice(void* host, const void* device, int size)
	{
		hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
	}

	//function to copy data from the host to the device
	void copyArrayToDevice(void* device, const void* host, int size)
	{
		hipMemcpy(device, host, size, hipMemcpyHostToDevice);
	}

	//function to copy data within the current device
	void copyArrayWithinDevice(void* deviceTo, const void* deviceFrom, int size)
	{
		hipMemcpy(deviceTo, deviceFrom, size, hipMemcpyDeviceToDevice);
	}
}
